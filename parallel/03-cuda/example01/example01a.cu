#include "hip/hip_runtime.h"
// =================================================================
//
// File: example01a.cu
// Author: Pedro Perez
// Description: This file implements the addition of two vectors 
//				using CUDA.
//              To compile:
//		        !nvcc -arch=sm_75 -o app example1a.cu
//
// Copyright (c) 2024 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.h"

using namespace std;
using namespace std::chrono;

#define SIZE 1000000000 // 1e9

__global__ void add_vector(int *result, int *a, int *b) {
    result[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(int argc, char* argv[]) {
    int *a, *b, *c;
    int *deviceA, *deviceB, *deviceC;

    // These variables are used to keep track of the execution time.
    high_resolution_clock::time_point start, end;
    double timeElapsed;

    a = new int [SIZE];
    b = new int [SIZE];
    c = new int [SIZE];

    hipMalloc((void**) &deviceA, SIZE * sizeof(int));
    hipMalloc((void**) &deviceB, SIZE * sizeof(int));
    hipMalloc((void**) &deviceC, SIZE * sizeof(int));

    fill_array(a, SIZE);
    display_array("a:", a);
    fill_array(b, SIZE);
    display_array("b:", b);

    hipMemcpy(deviceA, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

    cout << "Starting...\n";
    timeElapsed = 0;
    for (int j = 0; j < N; j++) {
        start = high_resolution_clock::now();

        add_vector<<<SIZE, 1>>>(deviceC, deviceA, deviceB);

        end = high_resolution_clock::now();
        timeElapsed += 
            duration<double, std::milli>(end - start).count();
    }
    hipMemcpy(c, deviceC, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    display_array("c:", c);
    cout << "avg time = " << fixed << setprecision(3) 
         << (timeElapsed / N) <<  " ms\n";

    delete [] a;
    delete [] b;
    delete [] c;

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}
