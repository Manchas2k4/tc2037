#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils/cheader.h"

#define SIZE 	1e6
#define THREADS 128

__global__ void add(int *a, int *b, int *c) {
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i < SIZE) {
		c[i] = a[i] + b[i];
	}
}

int main(int argc, char* argv[]) {
	int *a, *b, *c, i;
	int *d_a, *d_b, *d_c;
	double ms;
	
	a = (int*) malloc(SIZE * sizeof(int));
	fill_array(a, SIZE);
	display_array("a", a);
	
	b = (int*) malloc(SIZE * sizeof(int));
	fill_array(b, SIZE);
	display_array("b", b);
	
	c = (int*) malloc(SIZE * sizeof(int));

	hipMalloc((void**) &d_a, SIZE * sizeof(int));
	hipMalloc((void**) &d_b, SIZE * sizeof(int));
	hipMalloc((void**) &d_c, SIZE * sizeof(int));
	
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	
	ms = 0;
	for (i = 0; i < N; i++) {
		start_timer();
		add<<<SIZE/THREADS, THREADS>>>(d_a, d_b, d_c);
		ms += stop_timer();
	}
	
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	display_array("c", c);
	
	printf("avg time = %.15f\n", (ms / N));
	
	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);
	
	free(c);
	free(b);
	free(a);
	
	return 0;
}
